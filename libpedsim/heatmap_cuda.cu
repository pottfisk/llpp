#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "ped_model.h"

#include <stdio.h>


__global__ void increase_heat(Ped::Tagent* agents){
    int id = blockIdx.x*blockDim.x +threadIdx.x;
    agents[id].getDesiredX();
    agents[id].getDesiredY();
}

void intensify_heatmap(Ped::Tagent* agents){
    Ped::Tagent* a_device;
    hipMalloc(&a_device, sizeof(agents));
	hipMemcpy(a_device, &agents, sizeof(agents), hipMemcpyHostToDevice);
	increase_heat<<<1, agents.size()>>>(a_device);

}

int main(void){}
